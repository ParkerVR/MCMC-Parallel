#include "hip/hip_runtime.h"
// MARKOV CHAIN MONTE CARLO MAIN FILE

// Go to config or readme for compile instuctions


#include "mcmc-config.cuh" // The config can be swapped
#include "mcmc-headers.cuh" // Lists all internal includes




int main() {

  
  i_t sz = ARR_SZ;
  i_t lg = ARR_LG; 

  num_t* arr;
  arr = malloc(sizeof(num_t) * sz);


  #if SEEDED
    int seed = SEED;
  #else
    int seed = time(0);
    printf("\nSeed Generated: %d", seed);
  #endif

  i_t endstates = ENDSTATES;
  arr_init_cum_rand(arr, lg, endstates, seed);

  #if PRINT_ARR
    arr_print(arr, lg);
  #endif

  
  #if ENABLE_GPU
  
    time_g gpu_timer = gpu_timer_start();



    #if PRINT_TIME
      float gpu_time = gpu_time_elapsed(gpu_timer);
      printf("\nGPU Time Elapsed: %1.3f\n", gpu_time);
    #endif

  #endif
  

  #if ENABLE_SERIAL

    clock_t cpu_timer = cpu_timer_start();

    i_t outRow = mcmc_serial(arr, lg) - lg + endstates + 1;

    #if PRINT_RESULT
      printf("\n LG = %ld FINISHED AT OUTROW # %ld", lg, outRow);
    #endif

    #if PRINT_TIME
      float cpu_time = cpu_time_elapsed(cpu_timer);
      printf("\nCPU Time Elapsed: %1.3f\n", cpu_time);
    #endif

  #endif

  return 0;
}


// printf("%lu", sizeof(i_t)); // Tells num bytes in data type
// printf("%d", PTRDIFF_MAX); // Used to determine practical max bytes per array

